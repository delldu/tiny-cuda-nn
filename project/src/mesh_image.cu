#include "hip/hip_runtime.h"
/************************************************************************************
***
***     Copyright 2023 Dell Du(18588220928@163.com), All Rights Reserved.
***
***     File Author: Dell, 2023年 03月 07日 星期二 18:29:34 CST
***
************************************************************************************/
// #include "../include/mesh_image.h"
#include "../include/meshbox.h"
#include "../include/mesh_common.h"

#include <dirent.h>
#include <iostream>
#include <sys/stat.h> // dir
#include <sys/types.h>


GPUMemory<float> load_image(const std::string& filename, int& width, int& height)
{
    // width * height * RGBA
    float* out = load_stbi(&width, &height, filename.c_str());

    GPUMemory<float> result(width * height * 4);
    result.copy_from_host(out);
    free(out); // release memory of image data

    return result;
}

template <typename T>
__global__ void to_ldr(const uint64_t num_elements, const uint32_t n_channels,
    const uint32_t stride, const T* __restrict__ in, uint8_t* __restrict__ out)
{
    const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_elements)
        return;

    const uint64_t pixel = i / n_channels;
    const uint32_t channel = i - pixel * n_channels;

    out[i] = (uint8_t)(
        powf(fmaxf(fminf(in[pixel * stride + channel], 1.0f), 0.0f), 1.0f / 2.2f) * 255.0f + 0.5f);
}

template <typename T>
void save_image(const T* image, int width, int height, int n_channels, int channel_stride,
    const std::string& filename)
{
    GPUMemory<uint8_t> image_ldr(width * height * n_channels);
    linear_kernel(to_ldr<T>, 0, nullptr, width * height * n_channels, n_channels, channel_stride,
        image, image_ldr.data());

    std::vector<uint8_t> image_ldr_host(width * height * n_channels);
    CUDA_CHECK_THROW(hipMemcpy(
        image_ldr_host.data(), image_ldr.data(), image_ldr.size(), hipMemcpyDeviceToHost));

    save_stbi(image_ldr_host.data(), width, height, n_channels, filename.c_str());
}

// template <uint32_t stride>
// __global__ void eval_image(uint32_t n_elements, hipTextureObject_t texture,
//     float* __restrict__ xs_and_ys, float* __restrict__ result)
// {
//     uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i >= n_elements)
//         return;

//     uint32_t output_idx = i * stride;
//     uint32_t input_idx = i * 2;

//     float4 val = tex2D<float4>(texture, xs_and_ys[input_idx], xs_and_ys[input_idx + 1]);
//     result[output_idx + 0] = val.x;
//     result[output_idx + 1] = val.y;
//     result[output_idx + 2] = val.z;

//     for (uint32_t i = 3; i < stride; ++i) {
//         result[output_idx + i] = 1;
//     }
// }

vector<string> load_files(const string dirname, const string extname)
{
    DIR* dir;
    struct dirent* ent;
    std::vector<string> files;

    dir = opendir(dirname.c_str());
    if (dir == NULL) {
        tlog::error() << "Cannot open directory " << dirname;
        exit(EXIT_FAILURE);
    }

    while ((ent = readdir(dir)) != NULL) {
        char* name = ent->d_name;
        if (strcmp(name, ".") == 0 || strcmp(ent->d_name, "..") == 0)
            continue;

        if (strstr(name, extname.c_str()))
            files.push_back(dirname + "/" + string(name));
    }
    closedir(dir);

    return files;
}

void save_image_as_texture(GPUMemory<float> image, int width, int height, hipTextureObject_t texture)
{
    // int width, height;
    // GPUMemory<float> image = load_image(argv[1], width, height);

    // Create a cuda texture out of this image.
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = image.data();
    resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    resDesc.res.pitch2D.width = width;
    resDesc.res.pitch2D.height = height;
    resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = true;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

    CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, nullptr));
}

GPUMemory<float> load_image_and_depth(const std::string& image_filename,
    const std::string& depth_filename, int& width, int& height)
{
    // width * height * RGBA
    float* image_out = load_stbi(&width, &height, image_filename.c_str());

    int depth_width, depth_height;
    float* depth_out = load_stbi(&depth_width, &depth_height, depth_filename.c_str());
    if (width != depth_width || height != depth_height) {
        throw std::runtime_error{fmt::format("Image {} size is not same as depth {}", 
            image_filename, depth_filename)};
    }
    float *src = depth_out;
    float *dst = image_out;
    for (int i = 0; i < width * height; i++) {
        if (src[3] < 0.5f) { // Image masked, depth is far ...
            dst[3] = MAX_DEPTH;
        } else { // The feature of depth is more near, more bright
            dst[3] = (1.0f - src[0]) * 256.0f + (1.0f - src[1]) + (1.0f - src[2])/256.0f;
        }
        src += 4; dst += 4;
    }
    free(depth_out); // release memory of depth data

    GPUMemory<float> result(width * height * 4);
    result.copy_from_host(image_out);
    free(image_out); // release memory of image data

    return result;    
}
